
#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void P_chasing2(int *A, int iterations, int starting_index, long long int* t_value){  
           
           __shared__ long long int s_tvalue[1024 * 4];
           __shared__ int s_index[1024 * 4];
           int j = starting_index;
           
           long long int start_time = 0;
           long long int end_time = 0;
           long long int time_interval = 0;
           int it;


		    

           asm(".reg .u64 t1;\n\t"
           ".reg .u64 t2;\n\t");

           for (it = 0; it < iterations; it++){
                       asm("mul.wide.u32 t1, %2, %4;\n\t"          
                       "add.u64 t2, t1, %3;\n\t"            
                       "mov.u64 %0, %clock64;\n\t"                
                       "ld.global.u32 %1, [t2];\n\t"              
                       : "=l"(start_time), "=r"(j) : "r"(j), "l"(A), "r"(4));
                       
                       s_index[it] = j;
                       
                       asm volatile ("mov.u64 %0, %clock64;": "=l"(end_time));
                       
                       time_interval = end_time - start_time;
                       s_tvalue[it] = time_interval;
					   
           }
		   
		   
		     t_value[0] =  s_tvalue[it-1];
		   
		   
		   
		   
		   
}
