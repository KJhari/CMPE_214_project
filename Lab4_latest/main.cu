#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "kernel.cu"
#include <time.h>
#include "/usr/include/python2.7/Python.h"
#include "/home/nvidia/Documents/libxlsxwriter/include/xlsxwriter.h"

long int L1_Cache_size;
// suganya checking

void init_cpu_data(int *A, int size, int stride, int mod)
{
    for (int i = 0; i < size; i = i + stride)
    {
        A[i] = (i + stride) % mod;
    }
}
int check_cache_size()
{

    int ret = system("python ./parse_csv.py  --filename L1_Cache_Size_Calculation_new.csv --threshold 1000");

    return 0;
}

void get_L1_cache_size()
{
    FILE *fp;
    fp = fopen("output.txt", "r");
    if (fp == NULL) {
        printf("Error: Could not open file.\n");
    }

    // Read an integer value from the file and store it into a variable
    int value;
    if (fscanf(fp, "%ld", &L1_Cache_size) != 1) {
        printf("Error: Could not read integer value from file.\n");
        fclose(fp);
    }
     
    
    // Print the value to the console
    printf("Read integer value from file: %ld KB\n", (L1_Cache_size * 4)/1024);

    // Close the file
    fclose(fp);
    
}

void write_to_file(FILE *fp, long long int *t_value, int *A, int size)
{

    fprintf(fp, "%d", size);
    fputs(",", fp);
    fprintf(fp, "%llu", t_value[0]);
    fputs("\n", fp);

    // static int i=0,j=0;

    // lxw_workbook  *workbook  = workbook_new("L1_Cache_Size_Calculation.xlsx");
    // lxw_worksheet *worksheet = workbook_add_worksheet(workbook, NULL);

    // // Write some data to the worksheet
    // worksheet_write_number(worksheet, i, j, size, NULL);
    // worksheet_write_number(worksheet, i, j+1, t_value[0], NULL);
    // i++;

    // // Save the workbook and free memory
    // int status = workbook_close(workbook);
}

void L1_Cache_Size_compute()
{
    int *A_h;
    long long int *tvalue_h;
    int *A_d;
    long long int *tvalue_d;
    int array_size, mod, stride = 1;

    FILE *fp;
    fp = fopen("L1_Cache_Size_Calculation_new.csv", "a+");
    fputs("Array Size, Duration Time", fp);
    fputs("\n", fp);

    for (int array_size = 2; array_size < 131072 * 2; array_size *= 2)
    {
        mod = array_size;
        A_h = (int *)malloc(sizeof(int) * array_size);
        tvalue_h = (long long int *)malloc(sizeof(long long int) * array_size);

        hipDeviceSynchronize();
        hipMalloc((void **)&A_d, sizeof(int) * array_size);
        hipMalloc((void **)&tvalue_d, sizeof(long long int) * array_size);
        hipMemcpy(tvalue_d, tvalue_h, sizeof(long long int) * array_size, hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        init_cpu_data(A_h, array_size, stride, mod);

        hipDeviceSynchronize();

        hipMemcpy(A_d, A_h, sizeof(int) * array_size, hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        P_chasing2<<<1, 1>>>(A_d, array_size, 0, tvalue_d);
        hipDeviceSynchronize();

        hipMemcpy(tvalue_h, tvalue_d, sizeof(long long int) * array_size, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        write_to_file(fp, tvalue_h, A_h, array_size);
       

        hipDeviceSynchronize();

        free(A_h);
        free(tvalue_h);

        hipFree(A_d);
        hipFree(tvalue_d);
    }
    fclose(fp);
    check_cache_size();
    get_L1_cache_size();
}

int main(int argc, char *argv[])
{

    L1_Cache_Size_compute();

    return 0;
}
